#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_DEPRECATE
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <iostream>
#include "TreeNode.cuh"
#include "SeqBitmap.cuh"
#include "ResizableArray.h"
#include <fstream>
#include <map>
#include <stack>
#include <queue>
#include "GPUList.cuh"

using namespace std;
struct DbInfo{
	int cNum;
	int f1Size;
	DbInfo(int c, int f){
		f1Size = f;
		cNum = c;
	}
};

DbInfo ReadInput(char* input, float minSupPer, TreeNode **&f1, int *&index);
void IncArraySize(int*& array, int oldSize, int newSize);
int getBitmapType(int size);
void FindSeqPattern(stack<TreeNode*>*, int);
int MAX_WORK_SIZE;
int MAX_BLOCK_NUM;
int WORK_SIZE;
int MAX_THREAD_NUM;

int main(int argc, char** argv){

	// the input file name
	char * input = argv[1];
	// the minimun support in percentage
	float minSupPer = atof(argv[2]);

	MAX_BLOCK_NUM = 512;
	WORK_SIZE = MAX_BLOCK_NUM * 16;
	MAX_WORK_SIZE = MAX_BLOCK_NUM * 128;
	MAX_THREAD_NUM = 1024;

	SeqBitmap::memPos = false;
	TreeNode** f1 = NULL;
	int *index = NULL;
	stack<TreeNode*>* fStack = new stack<TreeNode*>;

	DbInfo dbInfo = ReadInput(input, minSupPer, f1, index);
	SList * f1List = new SList(dbInfo.f1Size);
	for (int i = 0; i < dbInfo.f1Size; i++){
		f1List->list[i] = i;
	}
	for (int i = 0; i < dbInfo.f1Size; i++){
		f1[i]->sList = f1List->get();
		f1[i]->iList = f1List->get();
		f1[i]->sListLen = dbInfo.f1Size;
		f1[i]->iListLen = dbInfo.f1Size - i - 1;
		f1[i]->iListStart = i + 1;
		f1[i]->iBitmap->CudaMemcpy();
	}

	for (int i = dbInfo.f1Size - 1; i >= 0; i--){
		fStack->push(f1[i]);
	}

	FindSeqPattern(fStack, minSupPer * dbInfo.cNum);

	delete f1List;
	delete fStack;
	delete [] index;
	delete [] f1;
	system("pause");

}

DbInfo ReadInput(char* input, float minSupPer, TreeNode  **&f1, int *&index){
	ResizableArray *cidArr = new ResizableArray(64);
	ResizableArray *tidArr = new ResizableArray(64);
	ResizableArray *iidArr = new ResizableArray(64);
	ifstream inFile;
	int custID;                   // current customer ID
	int transID;                  // current transaction ID
	int itemID;                   // current item ID
	int prevTransID = -1;         // previous transaction ID

	inFile.open(input);
	if (!inFile.is_open()){
		cout << "Cannot open file" << endl;
		exit(-1);
	}


	// initialize output variables
	int custCount = -1;               // # of customers in the dataset (largest ID)
	int itemCount = -1;               // # of items in the dataset (largest ID)
	int lineCount = 0;                // number of transaction
	int custTransSize = 400;
	int itemCustSize = 400;
	int *custTransCount = new int[custTransSize];
	int *itemCustCount = new int[itemCustSize];
	for (int i = 0; i < custTransSize; i++){
		custTransCount[i] = 0;
	}
	for (int i = 0; i < itemCustSize; i++){
		itemCustCount[i] = 0;
	}

	// this array stores the ID of the previous customer we have scanned and
	// has a certain item in his/her transactions.
	int *itemPrevCustID = new int[itemCustSize];
	for (int i = 0; i < itemCustSize; i++){
		itemPrevCustID[i] = -1;
	}

	while (!inFile.eof()){
		inFile >> custID;
		inFile >> transID;
		inFile >> itemID;

		// Copy the line of data into our resizable arrays
		cidArr->Add(custID);
		tidArr->Add(transID);
		iidArr->Add(itemID);

		// -- update the statistcs about customers
		if (custID >= custCount)
		{
			custCount = custID + 1;

			// make sure custTransCount is big enough
			if (custCount > custTransSize)
			{
				int newSize = (custCount > 2 * custTransSize) ?
				custCount : 2 * custTransSize;
				IncArraySize(custTransCount, custTransSize, newSize);
				custTransSize = newSize;
			}
			prevTransID = -1;
		}

		// increment custTransCount only if it's a different transaction
		if (prevTransID != transID)
		{
			custTransCount[custID]++;
			prevTransID = transID;
		}
		lineCount++;

		// -- update the statistics about items
		if (itemID >= itemCount)
		{
			itemCount = itemID + 1;

			// make sure itemCustCount is large enough
			if (itemCount >= itemCustSize)
			{
				int newSize = (itemCount > 2 * itemCustSize) ?
				itemCount : 2 * itemCustSize;
				IncArraySize(itemCustCount, itemCustSize, newSize);
				IncArraySize(itemPrevCustID, itemCustSize, newSize);
				itemCustSize = newSize;
			}
		}

		// update itemCustCount only if the item is from a different customer
		if (itemPrevCustID[itemID] != custID)
		{
			itemCustCount[itemID]++;
			itemPrevCustID[itemID] = custID;
		}
	}
	delete[] itemPrevCustID;
	inFile.close();

	// Copy the resizable array contents to the arrays containing
	// the in-memory cid/tid/iid lists
	int *cids, *tids, *iids;
	int overallCount;
	cidArr->ToArray(cids, overallCount);
	tidArr->ToArray(tids, overallCount);
	iidArr->ToArray(iids, overallCount);
	delete cidArr;
	delete tidArr;
	delete iidArr;

	cout << "custCount" << custCount << endl;
	cout << "itemCount" << itemCount << endl;
	cout << "minSup: " << float(custCount) * minSupPer << endl;
	int minSup = custCount * minSupPer;
	int f1Size = 0;
	map<int, int> f1map;
	ResizableArray *indexArray = new ResizableArray(10);
	for (int i = 0; i < itemCount; i++){
		if (itemCustCount[i] >= minSup) {
			(*indexArray).Add(i);
			f1map[i] = f1Size;
			f1Size++;
		}
	}
	cout << "f1Size: " << f1Size << endl;
	(*indexArray).ToArray(index, f1Size);
	delete indexArray;
	int maxCustTran = 0;
	int avgCustTran = 0;
	int sizeOfBitmaps[6] = { 0 };
	for (int i = 0; i < custCount; i++){
		if (custTransCount[i] > maxCustTran) maxCustTran = custTransCount[i];
		avgCustTran += custTransCount[i];
		sizeOfBitmaps[getBitmapType(custTransCount[i])]++;
	}
	if (maxCustTran > 64){
		cout << "A custumer has more than 64 transactions" << endl;
		exit(-1);
	}
	SeqBitmap::SetLength(sizeOfBitmaps[0], sizeOfBitmaps[1], sizeOfBitmaps[2], sizeOfBitmaps[3], sizeOfBitmaps[4]);
	cout << "Max number of transactions for a custumer is:" << maxCustTran << endl;
	cout << "total number of transactions is: " << avgCustTran << endl;
	cout << "Average number of transactions for a custumer is:" << avgCustTran / (custCount - 1) << endl;
	for (int i = 0; i < 6; i++){
		cout << "sizeOfBitmaps[" << i << "]: " << sizeOfBitmaps[i] << endl;
	}

	f1 = new TreeNode*[f1Size];
	for (int i = 0; i < f1Size; i++){
		f1[i] = new TreeNode;
		f1[i]->iBitmap = new SeqBitmap();
		f1[i]->iBitmap->Malloc();
		f1[i]->seq.push_back(index[i]);
		f1[i]->support = itemCustCount[index[i]];
	}
	TreeNode::f1 = f1;
	TreeNode::f1Len = f1Size;

	//index for different length bitmap
	int idx[5] = { 0 };
	int lastCid = -1;
	int lastTid = -1;
	int tidIdx = 0;
	int bitmapType;
	int current;
	cout << "OverallCount" << overallCount << endl;
	for (int i = 0; i < overallCount; i++){
		if (cids[i] != lastCid){
			lastCid = cids[i];
			bitmapType = getBitmapType(custTransCount[lastCid]);
			current = idx[bitmapType];
			idx[bitmapType]++;
			lastTid = tids[i];
			tidIdx = 0;
		}
		else if(tids[i] != lastTid){
			tidIdx++;
			lastTid = tids[i];
		}
		if (itemCustCount[iids[i]] >= minSup){
			f1[f1map[iids[i]]]->iBitmap->SetBit(bitmapType, current, tidIdx);
		}
	}
	delete [] cids;
	delete [] tids;
	delete [] iids;
	delete [] custTransCount;
	delete [] itemCustCount;
	return DbInfo(custCount, f1Size);
}

void IncArraySize(int*& array, int oldSize, int newSize)
{
	int i;

	// create a new array and copy data to the new one
	int *newArray = new int[newSize];
	for (i = 0; i < oldSize; i++)
		newArray[i] = array[i];
	for (i = oldSize; i < newSize; i++)
		newArray[i] = 0;

	// deallocate the old array and redirect the pointer to the new one
	delete[] array;
	array = newArray;
}

int getBitmapType(int size){
	if (size > 0 && size <= 4){
		return 0;
	}
	else if (size > 4 && size <= 8){
		return 1;
	}
	else if (size > 8 && size <= 16){
		return 2;
	}
	else if (size > 16 && size <= 32){
		return 3;
	}
	else if (size > 32 && size <= 64){
		return 4;
	}
	else{
		return 5;
	}
}

void FindSeqPattern(stack<TreeNode*>* fStack, int minSup){
	queue<TreeNode*> currentQueue;
	TreeNode* currentNodePtr;
	int sWorkSize = 0;
	int iWorkSize = 0;
	int sListLen;
	int iListLen;
	int iListStart;
	int *sResult = new int[MAX_WORK_SIZE];
	int * iResult = new int[MAX_WORK_SIZE];
	TreeNode ** sResultNodes = new TreeNode*[MAX_WORK_SIZE];
	TreeNode ** iResultNodes = new TreeNode*[MAX_WORK_SIZE];
	GPUList sgList[5] = { GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE) };
	GPUList igList[5] = { GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE), GPUList(MAX_WORK_SIZE) };
	for (int i = 0; i < 5; i++){
		sgList[i].result = sResult;
		igList[i].result = iResult;
	}
	while (!(fStack->empty())){
		cout << "fStack size: " << fStack->size() << endl;
		sWorkSize = 0;
		iWorkSize = 0;
		while (min(sWorkSize,iWorkSize) < WORK_SIZE || fStack->empty()){
			if (SeqBitmap::memPos){ 
				
			}
			else{
				currentNodePtr = fStack->top();
				sListLen = currentNodePtr->sListLen;
				iListLen = currentNodePtr->iListLen;
				iListStart = currentNodePtr->iListStart;
				if (sWorkSize + sListLen > MAX_WORK_SIZE || iWorkSize + currentNodePtr->iListLen > MAX_WORK_SIZE) break;
				for (int j = 0; j < sListLen; j++){
					//cout <<"j for sList: "<< j << endl;
					TreeNode* tempNode = new TreeNode;
					tempNode->iBitmap = new SeqBitmap();
					tempNode->iBitmap->CudaMalloc();
					tempNode->seq.push_back(NULL);
					tempNode->seq.push_back(currentNodePtr->sList->list[j]);
					sResultNodes[sWorkSize] = tempNode;
					sWorkSize++;
					for (int i = 0; i < 5; i++){
						sgList[i].AddToTail(currentNodePtr->iBitmap->gpuMemList[i], TreeNode::f1[currentNodePtr->sList->list[j]]->iBitmap->gpuMemList[i], tempNode->iBitmap->gpuMemList[i]);
					}
				}
				for (int j = 0; j < iListLen; j++){
					//cout << "j for iList: " << j << endl;
					TreeNode* tempNode = new TreeNode;
					tempNode->iBitmap = new SeqBitmap();
					tempNode->iBitmap->CudaMalloc();
					tempNode->seq.push_back(currentNodePtr->iList->list[j+iListStart]);
					iResultNodes[iWorkSize] = tempNode;
					iWorkSize++;
					for (int i = 0; i < 5; i++){
						igList[i].AddToTail(currentNodePtr->iBitmap->gpuMemList[i], TreeNode::f1[currentNodePtr->iList->list[j + iListStart]]->iBitmap->gpuMemList[i], tempNode->iBitmap->gpuMemList[i]);
					}
				}
				currentQueue.push(currentNodePtr);
				fStack->pop();
			}
		}
		if (SeqBitmap::memPos){

		}
		else{

			int *sgresult, *igresult;
			if (hipMalloc(&sgresult, sizeof(int)*sWorkSize) != hipSuccess){
				cout << "hipMalloc error in sgresult" << endl;
				exit(-1);
			}
			if (hipMemset(sgresult, 0, sizeof(int)*sWorkSize) != hipSuccess){
				cout << "hipMemset error in sgresult" << endl;
				exit(-1);
			}
			if (hipMalloc(&igresult, sizeof(int)*iWorkSize) != hipSuccess){
				cout << "hipMalloc error in igresult" << endl;
				exit(-1);
			}
			if (hipMemset(igresult, 0, sizeof(int)*iWorkSize) != hipSuccess){
				cout << "hipMemset error in igresult" << endl;
				exit(-1);
			}
			for (int i = 0; i < 5; i++){
				sgList[i].gresult = sgresult;
				igList[i].gresult = igresult;
				if (SeqBitmap::size[i] > 0){
					sgList[i].SupportCounting(MAX_BLOCK_NUM, MAX_THREAD_NUM, i, true);
					igList[i].SupportCounting(MAX_BLOCK_NUM, MAX_THREAD_NUM, i, false);
				}
			}
			for (int i = 0; i < 5; i++){
				if (SeqBitmap::size[i] > 0){
					sgList[i].clear();
					igList[i].clear();
				}
			}
			for (int i = 0; i < 5059; i++){
				if (sgList[0].result[i] != TreeNode::f1[i]->support){
					cout << "this should not happen" << endl;
					system("pause");
				}
			}

		}
		cout << "now we are here lol" << endl;
		system("pause");
	}
	delete [] sResultNodes;
	delete[] iResultNodes;
}